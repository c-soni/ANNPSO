#include "hip/hip_runtime.h"
#include "neural_network.h"
#define TILE_WIDTH 16
#define INF 1000000000.0f
//Basic cuda error checking macro
//TODO: Add cuRAND and cuBLAS error checking macros
//TODO: Wrap all calls in relevant error checking macros
#define cudaCheckError()\
{\
    hipError_t e = hipGetLastError();\
    if(e != hipSuccess)\
    {\
        printf("CUDA failure: %s%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e));\
        exit(EXIT_FAILURE);\
    }\
}

// Normalizes a vector of values in [0, 1] to [-MaxValue, MaxValue]
__global__
void Normalize(float *Array, int Number, float MaxValue)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Number)
        Array[Index] = 2 * (Array[Index] - 0.5f) * MaxValue;
}

// Transpose a matrix
__global__
void Transpose(float *InputMatrix, float *OutputMatrix, int Rows, int Columns)
{
    int IdX = blockDim.x * blockIdx.x + threadIdx.x;
    int IdY = blockDim.y * blockIdx.y + threadIdx.y;
    int TX = threadIdx.x;
    int TY = threadIdx.y;

    __shared__ float Tile[TILE_WIDTH][TILE_WIDTH];

    if(IdX < Columns && IdY < Rows)
    {
        Tile[TX][TY] = InputMatrix[IdX + Columns * IdY];
        OutputMatrix[IdY + Rows * IdX] = Tile[TX][TY];
    }
}

// Initialize to value
__global__
void InitToVal(float *Input, int Size, float Value)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
        Input[Index] = Value;
}

// Small kernel for device to device memory transfers
__global__
void DeviceToDevice(float *Destination, float *Source, int Size)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
        Destination[Index] = Source[Index];
}

// Initialize random number generator states
__global__
void InitRNGStates(hiprandState_t *States, int Size)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
        hiprand_init(Index, Index, Index, &States[Index]);
}

// LeakyReLU activation function
__global__
void LeakyReLU(float *Input, int Size)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
    {
        if(Input[Index] < 0.0f)
            Input[Index] = 0.001 * Input[Index];
    }
}

// Sigmoid activiation function
__global__
void Sigmoid(float *Input, int Size)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
        Input[Index] = (1 / (1 + __expf(-Input[Index])));
}

// SquaredError loss function
__global__
void SquaredError(float *Predicted, float *Actual, float *Fitness, int Size)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
        Fitness[Index] += (Predicted[Index] - (*Actual)) * (Predicted[Index] - (*Actual));
}

// Mean function
__global__
void Mean(float *Input, int NumElements, int Size)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Size)
        Input[Index] /= NumElements;
}

// Kernel which actually trains the data.
// __global__
// void FeedForward(NNParameters *NNP, PSOParameters *PSOP)
// {
//     int Index = blockDim.x * blockIdx.x + threadIdx.x;
//     __shared__ NNParameters NNParams;
//     __shared__ PSOParameters PSOParams;

//     if(threadIdx.x == 0)
//     {
//         NNParams = *NNP;
//         PSOParams = *PSOP;
//     }

//     if(Index < PSOParams.NumParticles)
//     {
//         //Pointer to weights and biases
//         float *WeightsAndBiases = &NNParams.WeightsAndBiases[Index * NNParams.NetworkSize];

//         //Input, output, matrix and temporary pointers
//         float *Input;
//         float *Output;
//         float *Matrix;
//         float *Temp;

//         //Fitness value
//         float Fitness = 0.0f;

//         //cuBLAS handle initialization
//         hipblasHandle_t Handle;
//         hipblasCreate(&Handle);

//         //Alpha and beta values
//         float Alpha = 1.0f;
//         float Beta = 0.0f;

//         Fitness = 0.0f;

//         //Main feed forward work to be done here
//         //Calculate fitness, i.e. loss (MSE?)
//         for(int j = 0; j < NNParams.NumVectors; j++)
//         {
//             //Input hidden multiplication + biases
//             Input = &(NNParams.InputFeatures[NNParams.InputNeurons * j]);
//             Output = &(NNParams.IntermediateIO[NNParams.MaxIOLength * Index]);
//             Matrix = &(NNParams.WeightsAndBiases[NNParams.NetworkSize * Index]);

//             hipblasSgemv(Handle, HIPBLAS_OP_N,
//                 NNParams.HiddenNeurons, NNParams.InputNeurons, &Alpha,
//                 Matrix, NNParams.HiddenNeurons, Input, 1, &Beta, Output, 1);
//             hipDeviceSynchronize();

//             Matrix += NNParams.InputNeurons * NNParams.HiddenNeurons;

//             //Add biases
//             hipblasSaxpy(Handle, NNParams.HiddenNeurons,
//                 &Alpha, Matrix, 1, Output, 1);

//             //Activation function
//             LeakyReLU <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.HiddenNeurons);
//             hipDeviceSynchronize();

//             Input = Output + NNParams.MaxIOLength / 2;
//             Matrix += NNParams.HiddenNeurons;

//             //Hidden hidden loop
//             for(int c = 1; c < NNParams.HiddenLayers; c++)
//             {
//                 //Swap input and output
//                 Temp = Input;
//                 Input = Output;
//                 Output = Temp;

//                 //Multiply
//                 hipblasSgemv(Handle, HIPBLAS_OP_N,
//                     NNParams.HiddenNeurons, NNParams.HiddenNeurons, &Alpha,
//                     Matrix, NNParams.HiddenNeurons, Input, 1, &Beta, Output, 1);
//                 hipDeviceSynchronize();

//                 Matrix += NNParams.HiddenNeurons * NNParams.HiddenNeurons;

//                 //Add biases
//                 hipblasSaxpy(Handle, NNParams.HiddenNeurons,
//                     &Alpha, Matrix, 1, Output, 1);

//                 //Activation function
//                 LeakyReLU <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.HiddenNeurons);
//                 hipDeviceSynchronize();

//                 Matrix += NNParams.HiddenNeurons;
//             }

//             //Hidden output multiplication + biases
//             //Multiply
//             hipblasSgemv(Handle, HIPBLAS_OP_N,
//                 NNParams.OutputNeurons, NNParams.HiddenNeurons, &Alpha,
//                 Matrix, NNParams.OutputNeurons, Input, 1, &Beta, Output, 1);
//             hipDeviceSynchronize();

//             Matrix += NNParams.HiddenNeurons * NNParams.OutputNeurons;

//             //Add biases
//             hipblasSaxpy(Handle, NNParams.OutputNeurons,
//                 &Alpha, Matrix, 1, Output, 1);

//             //Activation function
//             Sigmoid <<<(NNParams.OutputNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.OutputNeurons);
//             hipDeviceSynchronize();

//             Fitness += (NNParams.OutputFeatures[j] - Output[0]) * (NNParams.OutputFeatures[j] - Output[0]);
//         }

//         Fitness /= NNParams.NumVectors;
//         PSOParams.FitnessArray[Index] = Fitness;

//         //Ensure that no memory misalignment and access errors occur
//         hipblasDestroy(Handle);
//         //TODO: free any local memory at the end of the kernel
//     }
// }

// FeedForward function on CPU w/o cuBLAS Device API
void NeuralNetwork::FeedForward(NNParameters &NNParams, PSOParameters &PSOParams)
{
    //cuBLAS handle initialization
    hipblasHandle_t Handle;
    hipblasCreate(&Handle);

    //Alpha and beta values
    float Alpha = 1.0f;
    float Beta = 0.0f;

    //Input, output, matrix and temporary pointers
    float *Input;
    float *Output;
    float *Matrix;
    float *Temp;

    for(int j = 0; j < NNParams.NumVectors; j++)
    {
        for(int i = 0; i < PSOParams.NumParticles; i++)
        {
            Input = NNParams.InputFeatures + (NNParams.InputNeurons * j);
            Output = NNParams.IntermediateIO + (NNParams.MaxIOLength * i);
            Matrix = NNParams.WeightsAndBiases + (NNParams.NetworkSize * i);

            //Main feed forward work to be done here
            //Calculate fitness, i.e. loss (MSE?)

            //Input hidden multiplication + biases
            hipblasSgemv(Handle, HIPBLAS_OP_N,
                NNParams.HiddenNeurons, NNParams.InputNeurons, &Alpha,
                Matrix, NNParams.HiddenNeurons, Input, 1, &Beta, Output, 1);

            Matrix += NNParams.InputNeurons * NNParams.HiddenNeurons;

            //Add biases
            hipblasSaxpy(Handle, NNParams.HiddenNeurons,
                &Alpha, Matrix, 1, Output, 1);

            //Activation function
            LeakyReLU <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.HiddenNeurons);

            //Hidden hidden loop
            for(int c = 1; c < NNParams.HiddenLayers; c++)
            {
                //Swap input and output
                Temp = Input;
                Input = Output;
                Output = Temp;

                //Multiply
                hipblasSgemv(Handle, HIPBLAS_OP_N,
                    NNParams.HiddenNeurons, NNParams.HiddenNeurons, &Alpha,
                    Matrix, NNParams.HiddenNeurons, Input, 1, &Beta, Output, 1);

                Matrix += NNParams.HiddenNeurons * NNParams.HiddenNeurons;

                //Add biases
                hipblasSaxpy(Handle, NNParams.HiddenNeurons,
                    &Alpha, Matrix, 1, Output, 1);

                //Activation function
                LeakyReLU <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.HiddenNeurons);

                Matrix += NNParams.HiddenNeurons;
            }

            //Hidden output multiplication + biases
            //Multiply
            hipblasSgemv(Handle, HIPBLAS_OP_N,
                NNParams.OutputNeurons, NNParams.HiddenNeurons, &Alpha,
                Matrix, NNParams.OutputNeurons, Input, 1, &Beta, Output, 1);

            Matrix += NNParams.HiddenNeurons * NNParams.OutputNeurons;

            //Add biases
            hipblasSaxpy(Handle, NNParams.OutputNeurons,
                &Alpha, Matrix, 1, Output, 1);

            //Activation function
            Sigmoid <<<(NNParams.OutputNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.OutputNeurons);
            // MeanSquaredError <<<(NNParams.OutputNeurons - 1) / 32 + 1, 32>>> (PSOParams.FitnessArray, Output, NNParams.OutputNeurons);
        }

        //Calculate fitness
        float *OutputFeaturesPointer = NNParams.OutputFeatures + j;
        SquaredError <<<(PSOParams.NumParticles - 1) / 32 + 1, 32>>> (Output, OutputFeaturesPointer, PSOParams.FitnessArray, PSOParams.NumParticles);
    }

    // Calculate mean fitness
    Mean <<<(PSOParams.NumParticles - 1) / 32 + 1, 32>>> (PSOParams.FitnessArray, NNParams.NumVectors, PSOParams.NumParticles);

    //Ensure that no memory misalignment and access errors occur
    hipblasDestroy(Handle);
}

// PSO kernel
__global__
void PSO(NNParameters *NNP, PSOParameters *PSOP)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ NNParameters NNParams;
    __shared__ PSOParameters PSOParams;

    if(threadIdx.x == 0)
    {
        NNParams = *NNP;
        PSOParams = *PSOP;
    }

    if(Index < PSOParams.NumParticles)
    {
        //Initialize PBest, LBest and fitness
        float PersonalBest = PSOParams.PersonalBestFitness[Index];
        float PersonalBestX = INF;
        float LocalBestX = INF;
        int LocalBestIndex = Index;

        //Grid and block for network sized transfers
        dim3 NetworkGrid((NNParams.NetworkSize - 1) / 256 + 1, 1, 1);
        dim3 NetworkBlock(256, 1, 1);

        //Declare r1, r2
        float R1, R2;

        //Set left and right neighbours
        int Left = (PSOParams.NumParticles + Index - 1) % PSOParams.NumParticles;
        int Right = (1 + Index) % PSOParams.NumParticles;

        //Initialize random number generator states
        // hiprand_init(Index, Index, 0, &PSOParams.States[Index]);
        hiprandState_t LocalState = PSOParams.States[Index];

        //Pointer to weights and biases
        float *WeightsAndBiases = &NNParams.WeightsAndBiases[Index * NNParams.NetworkSize];
        float *PersonalBestWeights = &PSOParams.PersonalBestWeights[Index * NNParams.NetworkSize];

        //Load fitness value in local variable
        float Fitness = PSOParams.FitnessArray[Index];

        int Id = 0;

        //Compare fitness to personal best so far
        if(Fitness < PersonalBest)
        {
            //Copy personal best values
            PersonalBest = Fitness;
            PSOParams.PersonalBestFitness[Index] = Fitness;

            //Copy personal best weights and biases
            //Device to device transfer
            DeviceToDevice <<<NetworkGrid, NetworkBlock>>> (PersonalBestWeights, WeightsAndBiases, NNParams.NetworkSize);
            hipDeviceSynchronize();
        }
        __syncthreads();
        //Update local best particle index (left or right)
        if(PersonalBest > PSOParams.PersonalBestFitness[Left])
            LocalBestIndex = Left;
        if(PersonalBest > PSOParams.PersonalBestFitness[Right])
            LocalBestIndex = Right;
        __syncthreads();

        //Update weights and biases of each particle
        for (int i = 0; i < NNParams.NetworkSize; i++)
        {
            //Set index at which position needs to be updated
            Id = Index * NNParams.NetworkSize + i;

            //Set local best and personal best X (weights / biases)
            LocalBestX = PSOParams.PersonalBestWeights[LocalBestIndex * NNParams.NetworkSize + i];
            PersonalBestX = PSOParams.PersonalBestWeights[Index * NNParams.NetworkSize + i];

            //Generate random numbers
            R1 = hiprand_uniform(&LocalState);
            R2 = hiprand_uniform(&LocalState);

            //Update the velocity
            PSOParams.Velocities[Id] = PSOParams.Chi * (PSOParams.Velocities[Id] +
                                    PSOParams.C1 * R1 * (PersonalBestX - NNParams.WeightsAndBiases[Id]) +
                                    PSOParams.C2 * R2 * (LocalBestX - NNParams.WeightsAndBiases[Id]));

            //Ensure velocity values are within range
            // if (PSOParams.Velocities[Id] > PSOParams.VMax)
            //     PSOParams.Velocities[Id] = PSOParams.VMax;
            // if (PSOParams.Velocities[Id] < -PSOParams.VMax)
            //     PSOParams.Velocities[Id] = -PSOParams.VMax;

            //An interesting observation made today: not restricting the velocity
            //and instead only the position seems to yield much better results than
            //either restricting only the velocity or both or not restricting both

            __syncthreads();
            //Update the position
            NNParams.WeightsAndBiases[Id] = NNParams.WeightsAndBiases[Id] + PSOParams.Velocities[Id];

            // Ensure position values are within range
            if (NNParams.WeightsAndBiases[Id] > PSOParams.XMax)
            {
                NNParams.WeightsAndBiases[Id] = PSOParams.XMax;
                PSOParams.Velocities[Id] = 0.0f;
            }
            if (NNParams.WeightsAndBiases[Id] < -PSOParams.XMax)
            {
                NNParams.WeightsAndBiases[Id] = -PSOParams.XMax;
                PSOParams.Velocities[Id] = 0.0f;
            }
        }
        PSOParams.States[Index] = LocalState;
    }
}

void NeuralNetwork::CheckKernel()
{
    float *a = new float[12];
    float *b = new float[12];

    for(int i = 0; i < 3; i++)
    {
        for(int j = 0; j < 4; j++)
        {
            a[i * 4 + j] = i * 4 + j;
            std::cout << a[i * 4 + j] << " ";
        }
        std::cout << std::endl;
    }

    float *deva, *devb;
    hipMalloc((void**)&deva, 12 * sizeof(float));
    hipMalloc((void**)&devb, 12 * sizeof(float));

    hipMemcpy(deva, a, 12 * sizeof(float), hipMemcpyHostToDevice);
    dim3 Grid((4 - 1) / TILE_WIDTH + 1, (3 - 1) / TILE_WIDTH + 1, 1);
    dim3 Block(TILE_WIDTH, TILE_WIDTH, 1);
    Transpose <<<Grid, Block>>> (deva, devb, 3, 4);

    hipMemcpy(b, devb, 12 * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < 4; i++)
    {
        for(int j = 0; j < 3; j++)
        {
            std::cout << b[i * 3 + j] << " ";
        }
        std::cout << std::endl;
    }
}

//NeuralNetwork::NeuralNetwork()
// Constructor of the NeuralNetwork class
NeuralNetwork::NeuralNetwork(int InputNeurons, int HiddenLayers, int HiddenNeurons, int OutputNeurons, int NumParticles)
{
    //NN hyperparameters
    this->NNParams.InputNeurons = InputNeurons;
    this->NNParams.HiddenLayers = HiddenLayers;
    this->NNParams.HiddenNeurons = HiddenNeurons;
    this->NNParams.OutputNeurons = OutputNeurons;
    this->PSOParams.NumParticles = NumParticles;
    std::cout << "HYPERPARAMETERS SET" << std::endl;

    //Initialize random weights and biases on the GPU
    //Calculate total number of weights and biases for memory allocation
    int NetworkSize = ((InputNeurons + 1) * HiddenNeurons)
                                    + (((HiddenNeurons +1) * HiddenNeurons)
                                        * (HiddenLayers - 1))
                                    + ((HiddenNeurons + 1) * OutputNeurons);
    this->NNParams.NetworkSize = NetworkSize;

    //Total
    int TotalWeightsAndBiases = NumParticles * NetworkSize;

    std::cout << "TOTAL SPACE FOR WEIGHTS AND BIASES: " << TotalWeightsAndBiases * 4 / 1024 << "KB" << std::endl;

    //Allocate device memory for weights and biases
    float *WeightsAndBiases;
    hipMalloc((void**)&WeightsAndBiases, TotalWeightsAndBiases * sizeof(float));
    cudaCheckError();
    std::cout << "GPU SPACE ALLOCATED FOR WEIGHTS AND BIASES" << std::endl;

    //Allocate device memory for weights and biases
    float *PersonalBestWeights;
    hipMalloc((void**)&PersonalBestWeights, TotalWeightsAndBiases * sizeof(float));
    cudaCheckError();
    std::cout << "GPU SPACE ALLOCATED FOR PERSONAL BEST WEIGHTS AND BIASES" << std::endl;

    //Max space to be allocated to intermediate I/O
    int MaxIOLength = 2 * max(InputNeurons, max(HiddenNeurons, OutputNeurons));
    this->NNParams.MaxIOLength = MaxIOLength;
    float *IntermediateIO;
    hipMalloc((void**)&IntermediateIO, MaxIOLength * sizeof(float) * this->PSOParams.NumParticles);
    cudaCheckError();
    this->NNParams.IntermediateIO = IntermediateIO;

    //Allocate device memory for velocities
    float *Velocities;
    hipMalloc((void**)&Velocities, TotalWeightsAndBiases * sizeof(float));
    cudaCheckError();
    std::cout << "GPU SPACE ALLOCATED FOR VELOCITIES" << std::endl;

    //InitToVal grid and block
    dim3 InitGrid((this->PSOParams.NumParticles - 1) / 32 + 1, 1, 1);
    dim3 InitBlock(32, 1, 1);

    //Allocate device memory for fitness values
    float *FitnessArray;
    hipMalloc((void**)&FitnessArray, NumParticles * sizeof(float));
    cudaCheckError();
    InitToVal <<<InitGrid, InitBlock>>> (FitnessArray, this->PSOParams.NumParticles, 0.0f);
    cudaCheckError();
    this->PSOParams.FitnessArray = FitnessArray;
    std::cout << "GPU SPACE ALLOCATED FOR FITNESS VALUES" << std::endl;

    //Allocate device memory for fitness values
    float *PersonalBestFitness;
    hipMalloc((void**)&PersonalBestFitness, NumParticles * sizeof(float));
    cudaCheckError();
    InitToVal <<<InitGrid, InitBlock>>> (PersonalBestFitness, this->PSOParams.NumParticles, INF);
    cudaCheckError();
    this->PSOParams.PersonalBestFitness = PersonalBestFitness;
    std::cout << "GPU SPACE ALLOCATED FOR PERSONAL BEST FITNESS VALUES" << std::endl;

    //Initialize generator
    hiprandGenerator_t Gen;
    hiprandCreateGenerator(&Gen, HIPRAND_RNG_QUASI_SOBOL32);
    hiprandSetQuasiRandomGeneratorDimensions(Gen, this->NNParams.NetworkSize);
    hiprandSetPseudoRandomGeneratorSeed(Gen, time(NULL));
    std::cout << "CURAND GENERATOR INITIALIZED" << std::endl;

    //Dim3 variables for Normalize kernel
    dim3 NormalizeGrid(NetworkSize, 1, 1);
    dim3 NormalizeBlock(NumParticles, 1, 1);

    //Transpose grid and block
    dim3 TransposeGrid((this->PSOParams.NumParticles - 1) / TILE_WIDTH + 1, (this->NNParams.NetworkSize - 1) / TILE_WIDTH + 1, 1);
    dim3 TransposeBlock(TILE_WIDTH, TILE_WIDTH, 1);

    //Generate weights and biases
    hiprandGenerateUniform(Gen, WeightsAndBiases, TotalWeightsAndBiases);
    Normalize <<<NormalizeGrid, NormalizeBlock>>> (WeightsAndBiases, TotalWeightsAndBiases, 10.0f);
    cudaCheckError();
    Transpose <<<TransposeGrid, TransposeBlock>>> (WeightsAndBiases, PersonalBestWeights, this->NNParams.NetworkSize, this->PSOParams.NumParticles);
    cudaCheckError();
    this->NNParams.WeightsAndBiases = WeightsAndBiases;
    std::cout << "WEIGHTS AND BIASES INITIALIZED ON GPU" << std::endl;

    //Copy generated weights and biases to personal best array for initialization
    DeviceToDevice <<<NormalizeGrid, NormalizeBlock>>> (WeightsAndBiases, PersonalBestWeights, TotalWeightsAndBiases);
    this->PSOParams.PersonalBestWeights = PersonalBestWeights;

    //Generate velocities
    hiprandGenerateUniform(Gen, Velocities, TotalWeightsAndBiases);
    Normalize <<<NormalizeGrid, NormalizeBlock>>> (Velocities, TotalWeightsAndBiases, 1.0f);
    cudaCheckError();
    this->PSOParams.Velocities = Velocities;
    std::cout << "VELOCITIES INITIALIZED ON GPU" << std::endl;

    //Allocate space for hiprand states
    hiprandState_t *States;
    hipMalloc((void**)&States, NumParticles * sizeof(hiprandState_t));
    cudaCheckError();
    InitRNGStates <<<InitGrid, InitBlock>>> (States, this->PSOParams.NumParticles);
    cudaCheckError();
    this->PSOParams.States = States;
    std::cout << "SPACE ALLOCATED FOR CURAND STATES" << std::endl;

    //Synchronize all kernel calls upto this point
    hipDeviceSynchronize();
}

// NeuralNetwork::Load()
// Loads the input feature vectors into an array on the CPU and transfers it to
// the GPU. Method of transferring and thus training (with or without streams)
// will vary depending upon the size of input data.
void NeuralNetwork::Load(const char *FileName)
{
    int Size;
    float *InputFeatures;
    float *OutputFeatures;
    int InputWidth = this->NNParams.InputNeurons;
    int OutputWidth = this->NNParams.OutputNeurons;
    std::fstream FIn;
    FIn.open(FileName);
    if(!FIn.fail())
    {
        std::cout << "FILE OPENED" << std::endl;
        FIn >> Size;
        InputFeatures = new float[Size * InputWidth];
        OutputFeatures = new float[Size];
        std::cout << "SPACE ALLOCATED" << std::endl;
        int temp;

        for(int i = 0; i < Size; i++)
        {
            for(int j = 0; j < InputWidth; j++)
            {
                FIn >> temp;
                InputFeatures[i * InputWidth + j] = float(temp);
            }
            for(int j = 0; j < OutputWidth; j++)
            {
                FIn >> temp;
                OutputFeatures[i * OutputWidth + j] = float(temp);
            }
        }
    }
    FIn.close();

    std::cout << "INPUT OUTPUT SPACE REQUIRED: " << Size * 24 / 1024 << "KB" << std::endl;
    this->NNParams.NumVectors = Size;

    std::cout << "INPUT AND OUTPUT LOADED AND FILE CLOSED" << std::endl;

    //Transfer to GPU (Single hipMemcpy() for the time being)
    float* DeviceInputFeatures;
    hipMalloc((void**)&DeviceInputFeatures, Size * InputWidth * sizeof(float));
    cudaCheckError();
    hipMemcpy(DeviceInputFeatures, InputFeatures, Size * InputWidth * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError();
    this->NNParams.InputFeatures = DeviceInputFeatures;

    float* DeviceOutputFeatures;
    hipMalloc((void**)&DeviceOutputFeatures, Size * OutputWidth * sizeof(float));
    cudaCheckError();
    hipMemcpy(DeviceOutputFeatures, OutputFeatures, Size * OutputWidth * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError();
    this->NNParams.OutputFeatures = DeviceOutputFeatures;

    std::cout << "INPUT AND OUTPUT TRANSFERRED TO GPU" << std::endl;
}

// NeuralNetwork::Train()
// Trains the network using PSO and a set number of particles in order to eliminate
// backpropogation.
// Assumes weight matrix to be in column major format.
void NeuralNetwork::Train(int Epochs, const char *WeightsFile, bool Verbose)
{
    dim3 Grid((this->PSOParams.NumParticles - 1) / 32 + 1, 1, 1);
    dim3 Block(32, 1, 1);

    //NN parameters struct
    NNParameters NNParams;
    NNParams.Epochs = Epochs;
    NNParams.InputNeurons = this->NNParams.InputNeurons;
    NNParams.HiddenLayers = this->NNParams.HiddenLayers;
    NNParams.HiddenNeurons = this->NNParams.HiddenNeurons;
    NNParams.OutputNeurons = this->NNParams.OutputNeurons;
    NNParams.NetworkSize = this->NNParams.NetworkSize;
    NNParams.MaxIOLength = this->NNParams.MaxIOLength;
    NNParams.NumVectors = this->NNParams.NumVectors;
    NNParams.InputFeatures = this->NNParams.InputFeatures;
    NNParams.IntermediateIO = this->NNParams.IntermediateIO;
    NNParams.OutputFeatures = this->NNParams.OutputFeatures;
    NNParams.WeightsAndBiases = this->NNParams.WeightsAndBiases;

    //PSO parameters struct
    PSOParameters PSOParams;
    PSOParams.NumParticles = this->PSOParams.NumParticles;
    PSOParams.C1 = 2.05f;
    PSOParams.C2 = 2.05f;
    float Psi = PSOParams.C1 + PSOParams.C2;
    float Chi = abs(2.0f / (2.0f - Psi - sqrt(Psi * Psi - 4.0f * Psi)));
    PSOParams.Chi = Chi;
    PSOParams.XMax = 10.0f;
    PSOParams.VMax = 1.0f;
    PSOParams.FitnessArray = this->PSOParams.FitnessArray;
    PSOParams.PersonalBestFitness = this->PSOParams.PersonalBestFitness;
    PSOParams.States = this->PSOParams.States;
    PSOParams.PersonalBestWeights = this->PSOParams.PersonalBestWeights;
    PSOParams.Velocities = this->PSOParams.Velocities;

    NNParameters *D_NNParams;
    PSOParameters *D_PSOParams;

    hipMalloc((void**)&D_NNParams, sizeof(NNParameters));
    cudaCheckError();
    hipMalloc((void**)&D_PSOParams, sizeof(PSOParameters));
    cudaCheckError();

    hipMemcpy(D_NNParams, &NNParams, sizeof(NNParameters), hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemcpy(D_PSOParams, &PSOParams, sizeof(PSOParameters), hipMemcpyHostToDevice);
    cudaCheckError();

    float *Results = new float[this->PSOParams.NumParticles];
    int BestIndex = 0;
    float Best = INF;

    //Train using PSO
    for(int i = 0; i < Epochs; i++)
    {
        std::cout << "EPOCH (" << i + 1  << " / " << Epochs << ")" << std::endl;
        // FeedForward <<<Grid, Block>>> (D_NNParams, D_PSOParams);
        FeedForward(NNParams, PSOParams);
        hipDeviceSynchronize();
        cudaCheckError();
        PSO <<<Grid, Block>>> (D_NNParams, D_PSOParams);
        hipDeviceSynchronize();
        cudaCheckError();

        if(Verbose)
        {
            hipMemcpy(Results, PSOParams.PersonalBestFitness, this->PSOParams.NumParticles * sizeof(float), hipMemcpyDeviceToHost);
            cudaCheckError();
            BestIndex = 0;
            Best = Results[0];
            std::cout << "[" << Results[0];
            for(int j = 1; j < this->PSOParams.NumParticles; j++)
            {
                if(Best > Results[j])
                {
                    BestIndex = j;
                    Best = Results[j];
                }
                std::cout << ", " << Results[j];
            }
            std::cout << "]" << std::endl;
            std::cout << "BEST PARTICLE: " << BestIndex << std::endl;
            std::cout << "BEST FITNESS: " << Best << std::endl;
        }
    }

    hipMemcpy(Results, PSOParams.PersonalBestFitness, this->PSOParams.NumParticles * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError();

    BestIndex = 0;
    Best = Results[0];
    for(int i = 1; i < this->PSOParams.NumParticles; i++)
    {
        if(Best > Results[i])
        {
            BestIndex = i;
            Best = Results[i];
        }
    }

    std::cout << "FINAL BEST PARTICLE: " << BestIndex << std::endl;
    std::cout << "FINAL BEST FITNESS: " << Best << std::endl;

    float *DeviceBestNetwork = &this->PSOParams.PersonalBestWeights[this->NNParams.NetworkSize * BestIndex];
    float *BestNetwork = new float[this->NNParams.NetworkSize];
    hipMemcpy(BestNetwork, DeviceBestNetwork, this->NNParams.NetworkSize * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError();

    //Dump to file
    std::fstream FOut;
    FOut.open(WeightsFile, std::fstream::out);
    if(!FOut.fail())
    {
        FOut << this->NNParams.InputNeurons << std::endl;
        FOut << this->NNParams.HiddenLayers << std::endl;
        FOut << this->NNParams.HiddenNeurons << std::endl;
        FOut << this->NNParams.OutputNeurons << std::endl;
        for(int i = 0; i < this->NNParams.NetworkSize; i++)
        {
            FOut << BestNetwork[i] << std::endl;
        }
    }
    FOut.close();
}

// NeuralNetwork::Test()
// Tests a set of weights and biases and reports the loss
void NeuralNetwork::Test(const char *TestFile, const char *WeightsFile)
{
    std::fstream FIn;
    int InputNeurons = 0;
    int HiddenLayers = 0;
    int HiddenNeurons = 0;
    int OutputNeurons = 0;
    int NetworkSize = 0;
    float *Weights;
    FIn.open(WeightsFile, std::fstream::in);
    if(!FIn.fail())
    {
        FIn >> InputNeurons;
        FIn >> HiddenLayers;
        FIn >> HiddenNeurons;
        FIn >> OutputNeurons;

        NetworkSize = ((InputNeurons + 1) * HiddenNeurons)
                            + (((HiddenNeurons +1) * HiddenNeurons)
                                * (HiddenLayers - 1))
                            + ((HiddenNeurons + 1) * OutputNeurons);

        Weights = new float[NetworkSize];
        for(int i = 0; i < NetworkSize; i++)
            FIn >> Weights[i];
    }
    FIn.close();

    int NumSamples = 0;
    float *InputFeatures;
    float *OutputFeatures;
    FIn.open(TestFile, std::fstream::in);
    if(!FIn.fail())
    {
        FIn >> NumSamples;
        InputFeatures = new float[NumSamples * InputNeurons];
        OutputFeatures = new float[NumSamples * OutputNeurons];

        for(int i = 0; i < NumSamples; i++)
        {
            for(int j = 0; j < InputNeurons; j++)
                FIn >> InputFeatures[i * InputNeurons + j];

            for(int j = 0; j < OutputNeurons; j++)
                FIn >> OutputFeatures[i * OutputNeurons + j];
        }
    }
    FIn.close();

    float *InputVectors;
    hipMalloc((void**)&InputVectors, NumSamples * InputNeurons * sizeof(float));
    cudaCheckError();
    hipMemcpy(InputVectors, InputFeatures, NumSamples * InputNeurons * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError();

    float *WeightsAndBiases;
    hipMalloc((void**)&WeightsAndBiases, NetworkSize * OutputNeurons * sizeof(float));
    cudaCheckError();
    hipMemcpy(WeightsAndBiases, Weights, NetworkSize * OutputNeurons * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError();

    hipblasHandle_t Handle;
    hipblasCreate(&Handle);

    float Alpha = 1.0f, Beta = 0.0f;
    float Fitness = 0.0f, TempFitness = 0.0f;
    float *Input, *Output, *Matrix, *Temp;

    int MaxIOLength = 2 * max(InputNeurons, max(HiddenNeurons, OutputNeurons));
    float *IntermediateIO;
    hipMalloc((void**)&IntermediateIO, MaxIOLength * sizeof(float));
    cudaCheckError();

    //Main feed forward work to be done here
    //Calculate fitness, i.e. loss (MSE?)
    for(int j = 0; j < NumSamples; j++)
    {
        //Input hidden multiplication + biases
        Input = &InputVectors[InputNeurons * j];
        Output = IntermediateIO;
        Matrix = WeightsAndBiases;

        hipblasSgemv(Handle, HIPBLAS_OP_N,
            HiddenNeurons, InputNeurons, &Alpha,
            Matrix, HiddenNeurons, Input, 1, &Beta, Output, 1);

        Matrix += InputNeurons * HiddenNeurons;

        //Add biases
        hipblasSaxpy(Handle, HiddenNeurons,
            &Alpha, Matrix, 1, Output, 1);

        //Activation function
        LeakyReLU <<<(HiddenNeurons - 1) / 32 + 1, 32>>> (Output, HiddenNeurons);
        cudaCheckError();

        Input = Output + MaxIOLength / 2;
        Matrix += HiddenNeurons;

        //Hidden hidden loop
        for(int c = 1; c < HiddenLayers; c++)
        {
            //Swap input and output
            Temp = Input;
            Input = Output;
            Output = Temp;

            //Multiply
            hipblasSgemv(Handle, HIPBLAS_OP_N,
                HiddenNeurons, HiddenNeurons, &Alpha,
                Matrix, HiddenNeurons, Input, 1, &Beta, Output, 1);

            Matrix += HiddenNeurons * HiddenNeurons;

            //Add biases
            hipblasSaxpy(Handle, HiddenNeurons,
                &Alpha, Matrix, 1, Output, 1);

            //Activation function
            LeakyReLU <<<(HiddenNeurons - 1) / 32 + 1, 32>>> (Output, HiddenNeurons);
            cudaCheckError();

            Matrix += HiddenNeurons;
        }

        //Hidden output multiplication + biases
        //Multiply
        hipblasSgemv(Handle, HIPBLAS_OP_N,
            OutputNeurons, HiddenNeurons, &Alpha,
            Matrix, OutputNeurons, Input, 1, &Beta, Output, 1);

        Matrix += HiddenNeurons * OutputNeurons;

        //Add biases
        hipblasSaxpy(Handle, OutputNeurons,
            &Alpha, Matrix, 1, Output, 1);

        //Activation function
        Sigmoid <<<(OutputNeurons - 1) / 32 + 1, 32>>> (Output, OutputNeurons);
        cudaCheckError();

        hipMemcpy(&TempFitness, Output, OutputNeurons * sizeof(float), hipMemcpyDeviceToHost);
        Fitness += (OutputFeatures[j] - TempFitness) * (OutputFeatures[j] - TempFitness);
    }

    hipblasDestroy(Handle);
    Fitness /= NumSamples;

    std::cout << "TEST FITNESS: " << Fitness << std::endl;
}
